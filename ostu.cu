#include "hip/hip_runtime.h"
#include "func.h" // load image 格式
#include "func.cuh" // load image 格式
// image格式統一, 一維

__global__
void Zero(int* dst, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
	if ( index >= h*w ) {
        return;
    }
	dst[index] = 0;
}

__global__ float OtsuPart(uchar *dst, g, int *p, int fw, int fh) // (Mat hist)
{
	float avg1 = 1, avg2 = 1; // average1, average2
	int s1 = 0, s2 = 0; 
	
	int index = blockIdx.x *blockDim.x + threadIdx.x;
	if(index>=fh*fw){
		return;
	}
	
	int i = index % fw;
    int j = index / fw;
	
	for (int i = 0; i < img.w; i++) // i: row index
	{
		for (int j = 0; j < img.h; j++) // j: column index
		{
			int val = cvRound(dst.at<uchar>(i + j*img.h)); // 改成自己的版本	，給個數字可以知道像素，準備256*256
			if ( (i + j*img.h) < t)
			{
				s1 += (i + j*img.h)*val;
				avg1 += val;
			}
			else
			{
				s2 += (i + j*img.h)*val;
				avg2 += val;
			}
		}
	}
	
	float u1=0, u2=0;
	u1 = s1 / avg1;
	u2 = s2 / avg2;
	float sigma1 = 0, sigma2 = 0;
	
	for (int i = 0; i < img.w; i++) // i: row index
	{
		for (int j = 0; j < img.h; j++) // j: column index
		{
			int val = cvRound(dst.at<uchar>(i + j*img.h));
			if ( (i + j*img.h) < t)
			{
				sigma1 += (u1 - (i + j*img.h))*(u1 - (i + j*img.h))*val;
			}
			else
			{
				sigma2 += (u2 - (i + j*img.h))*(u2 - (i + j*img.h))*val;
			}
		}
	}
	
	return sigma1 + sigma2; // 
}

__global__
void copy_back(uchar* dst, int* src, int h, int w, int scaler)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    if ( index < h*w ) {
	    dst[index] = (uchar)( src[index] / scaler);
    }
}

void Otsu_compute(int* buffer, uchar* oldimg, int h, int w, int r)
{
	// memcpy(oldimg.pixel, img.pixel, (std::size_t) sizeof(uchar)*img.w * img.h ); // memory copying. 從img.pixel陣列複製長度為sizeof(uchar)*img.w * img.h(整張圖片長乘寬的數值)
	uchar* dst = img.pixel;
	
	float min = OtsuPart(img.pixel, 0);
	
	int index=0;
	int minindex = 256;
	//printf("min:%f", min);
	// 先改出C版本，再改成CUDA，
	// 格式統一
	
	for (int i = 0; i < img.w; i++) // i: row index
	{
		for (int j = 0; j < img.h; j++) // j: column index
		{
			dst[i + j*img.w] = 0; // initialization of dst[]
			
			float val = OtsuPart(img.pixel, i + j*img.w);
			printf("val:%f\n", val);
			if (val - min < 0)
			{
				min = val;
				minindex = i + j*img.w;
				dst[minindex]=min;
			}
		}
	}
	
	//return minindex; // 目的，找最小值
}

void Otsu(bitmap &img) // Mat hist
{
	hipError_t err; // 報錯用變數。
    uchar* pixel; // 整張圖，可以代表整張圖任一點的變數。
    uchar* oldpixel; // 舊圖，可以代表整張舊圖任一點的變數。
    int* buffer; // 緩衝, buffer，call by reference用。
    
	pixel = img.pixel; // 整張圖
    oldpixel = oldimg.pixel; // 整張舊圖
	
	err  = hipMalloc(&buffer, sizeof(int)*img.w*img.h);
    CHECK_ERROR(err)
	
	Zero<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(buffer, img.h, img.w);
    CHECK_LAST_ERROR
	Otsu_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(buffer, pixel, img.h, img.w, r);
    CHECK_LAST_ERROR

    copy_back<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, buffer, img.h, img.w, 1);
    CHECK_LAST_ERROR
	
	Zero<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(buffer, img.h, img.w);
    CHECK_LAST_ERROR
	Otsu_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(buffer, pixel, img.h, img.w, r);
    CHECK_LAST_ERROR

    copy_back<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, buffer, img.h, img.w, 1);
    CHECK_LAST_ERROR
	
	
}