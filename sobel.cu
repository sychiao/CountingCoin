#include "hip/hip_runtime.h"

#include "func.h"

static void Convolution(uchar *f, int *p, int fw, int fh, int *g, int gw, int gh)
{
	if (gw % 2 * gh % 2 == 0)
	{
        // Error
		return;
	}

	for (int i = 0; i < fw; i++)
	{
		for (int j = 0; j < fh; j++)
		{
			int val = 0;
			for (int k = 0; k < gw; k++)
			{
				for (int l = 0; l < gh; l++)
				{
					if (i - gw / 2 < 0 || j - gh / 2 < 0)
					{
						val += 0;
					}
					else
					{
						val += f[(i - gw / 2 + k) + (j - gh / 2 + l)*fw] * g[k + l*gw];
					}
				}
			}
			p[i + j*fw] = val;
		}
	}
}
void Sobel(bitmap &img)
{
	int a[] = { 1,0,-1,
		2,0,-2,
		1,0,-1 };
	int b[] = { 1,2,1,
		0,0,0,
		-1,-2,-1 };
	int bu[] = { 1,1,1,1,1,
		1,2,5,2,1,
		1,5,11,5,1,
		1,2,5,2,1,
		1,1,1,1,1 };

	int *bur = (int*)malloc(sizeof(int)*img.w*img.h);
    Convolution(img.pixel, bur, img.w, img.h, bu, 5, 5);

	for (int i = 0; i < img.w; i++)
	{
		for (int j = 0; j < img.h; j++)
		{
			img.pixel[i + j*img.h] = bur[i + j*img.h] / 55;
		}
	}

	int *gx = (int*)malloc(sizeof(int)*img.w*img.h);
	int *gy = (int*)malloc(sizeof(int)*img.w*img.h);

	Convolution(img.pixel, gx, img.w, img.h, a, 3, 3);
	Convolution(img.pixel, gy, img.w, img.h, b, 3, 3);

	for (int i = 0; i < img.w; i++)
	{
		for (int j = 0; j < img.h; j++)
		{
			int index = i + j*img.w;
			float G = sqrt(gx[index] * gx[index] + gy[index] * gy[index]);
			float theta;
			if (G > 75 && gx[index] != 0)
				theta = atan(gy[i + j*img.h] / gx[index]);
			else
				theta = 3.1415926 / 2;
			float dir = theta / (2 * 3.1415926);
			float value = 0;
			int x = 0, y = 0;
			if (dir< (1. / 8.) || dir>(7. / 8.))
				x = 1;
			else if (dir < (3. / 8.))
				y = -1;
			else if (dir < (5. / 8.))
				x = -1;
			else
				y = 1;
			index = (i + x) + (j + y)*img.w;
			if ((i + x) < img.w && (j + y) < img.h && (i + x)>0 && (j + y) >0)
			{
				if (sqrt(gx[index] * gx[index] + gy[index] * gy[index]) < G)
					value = G;
			}
			if (value > 75)
				img.pixel[i + j*img.w] = 255;
			else
				img.pixel[i + j*img.w] = 0;
		}
	}
}
