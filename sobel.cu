#include "hip/hip_runtime.h"

#include "func.h"
#define N 512



__global__
void Convolution(uchar *f, int *p, int fw, int fh, int a0b1bu2, int gw, int gh)
{
    // Share
    int a[] = { 1,0,-1,
                2,0,-2,
                1,0,-1 };
    int b[] = { 1,2,1,
                0,0,0,
               -1,-2,-1 };
	int bu[] = { 1,1,1,1,1,
		1,2,5,2,1,
		1,5,11,5,1,
		1,2,5,2,1,
		1,1,1,1,1 };
    int* select[] = {a,b,bu};
    int* g = select[a0b1bu2];
	if (gw % 2 * gh % 2 == 0)
	{
        // Error
        asm("trap;");
		return;
	}
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    if ( index >= fh*fw ) {
        return;
    }

    int i = index % fw;
    int j = index / fw;

    int val = 0;
    for (int k = 0; k < gw; k++)
    {
        for (int l = 0; l < gh; l++)
        {
            if (i - gw / 2 < 0 || j - gh / 2 < 0)
            {
                val += 0;
            }
            else
            {
                val += f[(i - gw / 2 + k) + (j - gh / 2 + l)*fw] * g[k + l*gw];
            }
        }
    }
    p[i + j*fw] = val;
}

__global__
void copy_back(uchar* dst, int* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    if ( index < h*w ) {
	    dst[index] = src[index] / 55;
    }
}


__global__
void compute(uchar* pixel, int* gx, int* gy, int w, int h)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    if ( index >= h*w ) {
        return;
    }

    int i = index % w;
    int j = index / w;

    float G = sqrt((float)gx[index] * gx[index] + gy[index] * gy[index]);
    float theta;
    if (G > 75 && gx[index] != 0)
        theta = atan((float) gy[i + j*h] / gx[index]);
    else
        theta = 3.1415926 / 2;
    float dir = theta / (2 * 3.1415926);
    float value = 0;
    int x = 0, y = 0;
    if (dir< (1. / 8.) || dir>(7. / 8.))
        x = 1;
    else if (dir < (3. / 8.))
        y = -1;
    else if (dir < (5. / 8.))
        x = -1;
    else
        y = 1;
    index = (i + x) + (j + y)*w;
    if ((i + x) < w && (j + y) < h && (i + x)>0 && (j + y) >0)
    {
        if (sqrt((float) gx[index] * gx[index] + gy[index] * gy[index]) < G)
            value = G;
    }
    if (value > 75)
        pixel[i + j*w] = 255;
    else
        pixel[i + j*w] = 0;
}

void Sobel(bitmap &img)
{
/*
	int *bur = (int*)malloc(sizeof(int)*img.w*img.h);
	int *gx = (int*)malloc(sizeof(int)*img.w*img.h);
	int *gy = (int*)malloc(sizeof(int)*img.w*img.h);
*/
	int *bur;
	int *gx;
	int *gy;
    uchar* pixel;

// Not support full cuda
//*
    
    hipMalloc(&pixel, sizeof(int)*img.w*img.h);
    hipMemcpy(pixel, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
//*/

    hipMalloc(&bur, sizeof(int)*img.w*img.h);
    hipMalloc(&gx, sizeof(int)*img.w*img.h);
    hipMalloc(&gy, sizeof(int)*img.w*img.h);

    Convolution<<<(img.w * img.h + N) / N, N>>>(pixel, bur, img.w, img.h, 2, 5, 5);

    // Kernel 1
    copy_back<<<(img.w * img.h + N) / N, N>>>(pixel, bur, img.h, img.w);

	Convolution<<<(img.w * img.h + N) / N, N>>>(pixel, gx, img.w, img.h, 0, 3, 3);
	Convolution<<<(img.w * img.h + N) / N, N>>>(pixel, gy, img.w, img.h, 1, 3, 3);

    // Kernel 2
    compute<<<(img.w * img.h + N) / N, N>>>(pixel, gx, gy, img.w, img.h);

    hipMemcpy(img.pixel, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);

    hipFree(bur);
    hipFree(gx);
    hipFree(gy);
    hipFree(pixel);
}
