#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include "func.h"
#include "func.cuh"

using namespace cv;

int main()
{
    struct timeval tv0, tv;
    gettimeofday(&tv0,NULL);

	Mat srcImg = imread("coincoin.png", CV_LOAD_IMAGE_GRAYSCALE);

	threshold(srcImg, srcImg, 190, 255, THRESH_BINARY | THRESH_OTSU);
	Mat mask1 = getStructuringElement(MORPH_RECT, Size(6, 6));

    Mat n = srcImg.clone();
	bitmap img(srcImg);
    bitmap oldimg(img);

    uchar* tmp1;
    uchar* tmp2;
    uchar* d_img_pixel;
    uchar* d_oldimg_pixel;
    uchar* d_buffer;
    hipMalloc(&d_img_pixel, sizeof(int) * img.w * img.h);
    hipMalloc(&d_oldimg_pixel, sizeof(uchar) * img.w * img.h);
    hipMalloc(&d_buffer, sizeof(int) * img.w * img.h);

    /************ 8 Line *****************************/
    hipMemcpy(d_img_pixel, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
    hipMemcpy(d_oldimg_pixel, oldimg.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
    tmp1 = img.pixel;
    tmp2 = oldimg.pixel;
    img.pixel = d_img_pixel;
    oldimg.pixel = d_oldimg_pixel;
    /****************************************/

        TimeDiff(&tv0,&tv);
	Erode(img, d_buffer);
        TimeDiff(&tv0,&tv);
	Dilate(img, d_buffer);
        TimeDiff(&tv0,&tv);


    /********* 5 Linw img show ***************/
//    hipMemcpy(tmp1, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
//    n.data = tmp1;
//	imshow("Tmp", n);
    /***********************************/

	Sobel(img,d_buffer);
        TimeDiff(&tv0,&tv);

    oldimg.pixel = (uchar*)malloc(sizeof(uchar)*img.w*img.h);

   // for(int r=50;r<150;r+=2)
    Hough(img, oldimg, 113, d_buffer);
        TimeDiff(&tv0,&tv);

    /************ 8 Line *****************************/
    hipMemcpy(tmp1, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    hipMemcpy(tmp2, oldimg.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    img.pixel = tmp1;
    oldimg.pixel = tmp2;
    /****************************************/

	Mat m2 = Mat(img.h, img.w, CV_8UC1);
	memcpy(m2.data, img.pixel, img.w*img.h * sizeof(uchar));
//	imshow("After hough", m2);
	GaussianBlur(m2, m2, Size(9, 9), 0, 0);
	threshold(m2, m2, 30, 255, THRESH_BINARY );

    hipFree(d_img_pixel);
    hipFree(d_oldimg_pixel);
    hipFree(d_buffer);
	//imshow("Final", m2);
	//waitKey(0);

        TimeDiff(&tv0,&tv);
	return 0;
}
