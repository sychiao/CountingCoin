#include "hip/hip_runtime.h"
#include "func.h"


__global__
void Zero(uchar* dst, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
	if ( index >= h*w ) {
        return;
    }
	dst[index] = 0;
}

__global__
void Hough_compute(uchar* img, uchar* oldimg, int h, int w, int r)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    uchar* dst = img;
	if ( index >= h*w ) {
        return;
    }
	int i = index % w;
    int j = index / w;

#define ONE_STEP 10

	if ((oldimg[index] > 0) && (i % ONE_STEP == 0) && (j % ONE_STEP == 0) ) {
		dst[index] = 190;
		//int a = 0;
        int x, y;
		for (int t = 0; t < 500; t++)
		{
			x = i + r * cos(3.1415926 / 250 * t);
			y = j + r * sin(3.1415926 / 250 * t);
			if (x >= 0 && x < w && y >= 0 && y < h) {
				atomicAdd((int*)&dst[x + y * w], 30);
            }
		}
	}
}
void Hough(bitmap &img, bitmap &oldimg, int r)
{
    hipError_t err;
//* Not support full cuda
    uchar* pixel;
    uchar* oldpixel;

    err = hipMalloc(&pixel, sizeof(uchar)*img.w*img.h);
    CHECK_ERROR( err)
    err = hipMalloc(&oldpixel, sizeof(uchar)*img.w*img.h);
    CHECK_ERROR( err)

    err= hipMemcpy(pixel, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
    CHECK_ERROR( err)
    //hipMemcpy(oldpixel, oldimg.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
//*/

    //memcpy( oldimg.pixel, img.pixel, (std::size_t) sizeof(uchar)*img.w * img.h );
    err= hipMemcpy(oldpixel, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToDevice);
    CHECK_ERROR( err)

    Zero<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, img.h, img.w);
    CHECK_LAST_ERROR
	Hough_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, oldpixel, img.h, img.w, r);
    CHECK_LAST_ERROR
//*

    err= hipMemcpy(img.pixel, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    CHECK_ERROR( err)
    err= hipMemcpy(oldimg.pixel, oldpixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    CHECK_ERROR( err)
    err= hipFree(pixel);
    CHECK_ERROR( err)
    err= hipFree(oldpixel);
    CHECK_ERROR( err)

//*/
}
