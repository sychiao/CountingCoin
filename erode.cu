#include "hip/hip_runtime.h"
#include "func.h"
/**int getindex(int x, int y)
{
	if (x > w || y > h || x < 0 || y<0)
		return 0;
	return x + y*w;
}*/

#define getindex( x,y) ( x) + ( y) * w

__global__
void Dilate_compute(uchar* dst, uchar* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int i = index % w;
    int j = index / w;

    if ( index >= h * w ) {
        return;
    }

	if ( i <= 3 && i >= w-3 && j <= 3 && j >= h-3 )
	{
        dst[index] = src[index];
        return;
    }

    int start = 0;
    for (int k = -3; k <= 3; k++)
    {
        for (int l = -3; l <= 3; l++)
        {
            if (src[getindex(i + k, j + l)] == 255)
            {
                dst[index] = 255;
                start = 1;
                break;
            }
        }
        if (start == 1)
            break;
    }
    if(start == 0)
        dst[index] = 0;
}

void Dilate(bitmap &img)
{
    uchar* pixel = img.pixel;
    uchar* ptr;

    hipMalloc(&ptr, sizeof(uchar)*img.w*img.h);
//    hipMemcpy(ptr, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToDevice);


	//uchar *ptr = (uchar*)malloc(sizeof(uchar)*img.w*img.h);
	//memcpy(ptr, img.pixel, sizeof(uchar)*img.h*img.w);

    Dilate_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(ptr, pixel, img.h, img.w);

    img.pixel = ptr;
    ptr = pixel;
    hipFree(ptr);
}


__global__
void Erode_compute(uchar* dst, uchar* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int i = index % w;
    int j = index / w;

    if ( index >= h * w ) {
        return;
    }

	if ( i <= 3 && i >= w-3 && j <= 3 && j >= h-3 )
	{
        dst[index] = src[index];
        return;
    }


    int start = 0;
    for (int k = -3; k <= 3; k++)
    {
        for (int l = -3; l <= 3; l++)
        {
            if (src[getindex(i + k, j + l)] != 255)
            {
                dst[index] = 0;
                start = 1;
                break;
            }
        }
        if (start == 1)
            break;
    }
    if (start == 0)
        dst[index] = 255;

}


void Erode(bitmap &img)
{
    uchar* pixel = img.pixel;
    uchar* ptr;

    hipMalloc(&ptr, sizeof(uchar)*img.w*img.h);
//    hipMemcpy(ptr, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToDevice);

    Erode_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(ptr, pixel, img.h, img.w);

    img.pixel = ptr;
    ptr = pixel;
    hipFree(ptr);
}
