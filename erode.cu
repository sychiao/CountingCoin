#include "hip/hip_runtime.h"
#include "func.h"
/**int getindex(int x, int y)
{
	if (x > w || y > h || x < 0 || y<0)
		return 0;
	return x + y*w;
}*/

#define getindex( x,y) ( x) + ( y) * w

__global__
void Dilate_compute(uchar* dst, uchar* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int i = index % w;
    int j = index / w;

	if ( i <= 3 && i >= w-3 && j <= 3 && j >= h-3 )
	{
        return;
    }

    int start = 0;
    for (int k = -3; k <= 3; k++)
    {
        for (int l = -3; l <= 3; l++)
        {
            if (src[getindex(i + k, j + l)] == 255)
            {
                dst[index] = 255;
                start = 1;
                break;
            }
        }
        if (start == 1)
            break;
    }
    if(start == 0)
        dst[index] = 0;
}

void Dilate(bitmap &img)
{
    uchar* pixel = img.pixel;
    uchar* ptr;

    hipMalloc(&ptr, sizeof(uchar)*img.w*img.h);
    hipMemcpy(ptr, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToDevice);


	//uchar *ptr = (uchar*)malloc(sizeof(uchar)*img.w*img.h);
	//memcpy(ptr, img.pixel, sizeof(uchar)*img.h*img.w);

    Dilate_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, ptr, img.h, img.w);


    hipFree(ptr);
}


__global__
void Erode_compute(uchar* dst, uchar* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int i = index % w;
    int j = index / w;

	if ( i <= 3 && i >= w-3 && j <= 3 && j >= h-3 )
	{
        return;
    }


    int start = 0;
    for (int k = -3; k <= 3; k++)
    {
        for (int l = -3; l <= 3; l++)
        {
            if (src[getindex(i + k, j + l)] != 255)
            {
                dst[index] = 0;
                start = 1;
                break;
            }
        }
        if (start == 1)
            break;
    }
    if (start == 0)
        dst[index] = 255;

}


void Erode(bitmap &img)
{
    uchar* pixel = img.pixel;
    uchar* ptr;

    hipMalloc(&ptr, sizeof(uchar)*img.w*img.h);
    hipMemcpy(ptr, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToDevice);

    Erode_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, ptr, img.h, img.w);

    hipFree(ptr);
}
