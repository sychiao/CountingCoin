#include "hip/hip_runtime.h"
#include "func.h"
/**int getindex(int x, int y)
{
	if (x > w || y > h || x < 0 || y<0)
		return 0;
	return x + y*w;
}*/

#define getindex( x,y) ( x) + ( y) * w

__global__
void Dilate_compute(uchar* dst, uchar* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int i = index % w;
    int j = index / w;

	if ( i <= 3 && i >= w-3 && j <= 3 && j >= h-3 )
	{
        return;
    }

    int start = 0;
    for (int k = -3; k <= 3; k++)
    {
        for (int l = -3; l <= 3; l++)
        {
            if (src[getindex(i + k, j + l)] == 255)
            {
                dst[index] = 255;
                start = 1;
                break;
            }
        }
        if (start == 1)
            break;
    }
    if(start == 0)
        dst[index] = 0;
}

void Dilate(bitmap &img)
{
    //*
        uchar* pixel;
        uchar* ptr;

        hipMalloc(&pixel, sizeof(uchar)*img.w*img.h);
        hipMemcpy(pixel, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);

        hipMalloc(&ptr, sizeof(uchar)*img.w*img.h);
        hipMemcpy(ptr, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToDevice);


    //*/
	//uchar *ptr = (uchar*)malloc(sizeof(uchar)*img.w*img.h);
	//memcpy(ptr, img.pixel, sizeof(uchar)*img.h*img.w);

    Dilate_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, ptr, img.h, img.w);

//	memcpy(img.pixel, ptr, sizeof(uchar)*img.h*img.w);
//    free(img.pixel);
    hipMemcpy(img.pixel, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    hipFree(pixel);
    hipFree(ptr);
    //img.pixel = ptr;
}

void Erode(bitmap &img)
{
    int w = img.w;
	uchar *ptr = (uchar*)malloc(sizeof(uchar)*img.w*img.h);
	memcpy(ptr, img.pixel, sizeof(uchar)*img.h*img.w);	
	for (int i = 3; i < img.w - 3; i++)
	{
		for (int j = 3; j < img.h - 3; j++)
		{
			int start = 0;
            int index = i + j * img.w;
			for (int k = -3; k <= 3; k++)
			{
				for (int l = -3; l <= 3; l++)
				{
					if (img.pixel[getindex(i + k, j + l)] != 255)
					{
						ptr[index] = 0;
						start = 1;
						break;
					}
				}
				if (start == 1)
					break;
			}
			if (start == 0)
				ptr[index] = 255;
				
		}
	}
	//memcpy(img.pixel, ptr, sizeof(uchar)*img.h*img.w);	
    free(img.pixel);
    img.pixel = ptr;
}
