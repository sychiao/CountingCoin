#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <math.h>
#include "func.h"
#include "func.cuh"

using namespace cv;

// global tmp memory
//__device__ int d_tmp[10];

int main()
{
    struct timeval tv0, tv;
    gettimeofday(&tv0,NULL);
        gettimeofday(&tv,NULL);
        TimeDiff(tv0,tv);


	Mat srcImg = imread("coincoin.png", CV_LOAD_IMAGE_GRAYSCALE);


// Prepare global tmp mem
//    uchar** tmp;
//    hipError_t err;
//    size_t s;
//    err = hipGetSymbolSize( &s, HIP_SYMBOL("d_tmp"
//            ))   ;
    //err = hipGetSymbolAddress((void**)&tmp,"d_tmp");
//    CHECK_ERROR(err);

	threshold(srcImg, srcImg, 190, 255, THRESH_BINARY | THRESH_OTSU);
	Mat mask1 = getStructuringElement(MORPH_RECT, Size(6, 6));

    Mat n = srcImg.clone();
	bitmap img(srcImg);
    bitmap oldimg(img);

    uchar* tmp1;
    uchar* tmp2;
    uchar* d_img_pixel;
    uchar* d_oldimg_pixel;
    hipMalloc(&d_img_pixel, sizeof(uchar) * img.w * img.h);
    hipMalloc(&d_oldimg_pixel, sizeof(uchar) * img.w * img.h);

    /************ 8 Line *****************************/
    hipMemcpy(d_img_pixel, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
    hipMemcpy(d_oldimg_pixel, oldimg.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
    tmp1 = img.pixel;
    tmp2 = oldimg.pixel;
    img.pixel = d_img_pixel;
    oldimg.pixel = d_oldimg_pixel;
    /****************************************/

        gettimeofday(&tv,NULL);
        TimeDiff(tv0,tv);
	Erode(img);
        gettimeofday(&tv,NULL);
        TimeDiff(tv0,tv);
	Dilate(img);


    /********* 5 Linw img show ***************/
//    hipMemcpy(tmp1, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
//    n.data = tmp1;
//	imshow("Tmp", n);
    /***********************************/

        gettimeofday(&tv,NULL);
        TimeDiff(tv0,tv);
	Sobel(img);

    oldimg.pixel = (uchar*)malloc(sizeof(uchar)*img.w*img.h);

   // for(int r=50;r<150;r+=2)
        gettimeofday(&tv,NULL);
        TimeDiff(tv0,tv);
    Hough(img, oldimg, 113);
        gettimeofday(&tv,NULL);
        TimeDiff(tv0,tv);

    /************ 8 Line *****************************/
    hipMemcpy(tmp1, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    hipMemcpy(tmp2, oldimg.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    img.pixel = tmp1;
    oldimg.pixel = tmp2;
    /****************************************/

	Mat m2 = Mat(img.h, img.w, CV_8UC1);
	memcpy(m2.data, img.pixel, img.w*img.h * sizeof(uchar));
//	imshow("After hough", m2);
	GaussianBlur(m2, m2, Size(9, 9), 0, 0);
	threshold(m2, m2, 30, 255, THRESH_BINARY );

	//imshow("Final", m2);
	//waitKey(0);
	return 0;
}
