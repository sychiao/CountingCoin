#include "hip/hip_runtime.h"
#include "func.h"
#include "func.cuh"


__global__
void Zero(int* dst, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
	if ( index >= h*w ) {
        return;
    }
	dst[index] = 0;
}

__global__
void Hough_compute(int* buffer, uchar* oldimg, int h, int w, int r)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int* dst = buffer;
	if ( index >= h*w ) {
        return;
    }
	int i = index % w;
    int j = index / w;

#define ONE_STEP 10

	if ((oldimg[index] > 0) && (i % ONE_STEP == 0) && (j % ONE_STEP == 0) ) {
		dst[index] = 190;
		//int a = 0;
        int x, y;
		for (int t = 0; t < 500; t++)
		{
			x = i + r * cos(3.1415926 / 250 * t);
			y = j + r * sin(3.1415926 / 250 * t);
			if (x >= 0 && x < w && y >= 0 && y < h) {
				atomicAdd((int*)&dst[x + y * w], 30);
            }
		}
	}

}
void Hough(bitmap &img, bitmap &oldimg, int r, uchar* &d_buffer)
{
    hipError_t err;
    uchar* pixel;
    int* buffer = (int*)d_buffer;
    pixel = img.pixel;
/* Not support full cuda

    err = hipMalloc(&pixel, sizeof(uchar)*img.w*img.h);
    CHECK_ERROR( err)
    err = hipMalloc(&oldpixel, sizeof(uchar)*img.w*img.h);
    CHECK_ERROR( err)

    err= hipMemcpy(pixel, img.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
    CHECK_ERROR( err)
    
    memcpy( oldimg.pixel, img.pixel, (std::size_t) sizeof(uchar)*img.w * img.h );
    err= hipMemcpy(oldpixel, oldimg.pixel, sizeof(uchar)*img.w*img.h, hipMemcpyHostToDevice);
    CHECK_ERROR( err)
*/


//    err= hipMemcpy(oldpixel, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToDevice);
//    CHECK_ERROR( err)

    Zero<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(buffer, img.h, img.w);
    CHECK_LAST_ERROR
	Hough_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(buffer, pixel, img.h, img.w, r);
    CHECK_LAST_ERROR

    copy_back<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(pixel, buffer, img.h, img.w, 1);
    CHECK_LAST_ERROR

/*

    err= hipMemcpy(img.pixel, pixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    CHECK_ERROR( err)
    err= hipMemcpy(oldimg.pixel, oldpixel, sizeof(uchar)*img.w*img.h, hipMemcpyDeviceToHost);
    CHECK_ERROR( err)
    err= hipFree(pixel);
    CHECK_ERROR( err)
    err= hipFree(oldpixel);
    CHECK_ERROR( err)

*/
}
