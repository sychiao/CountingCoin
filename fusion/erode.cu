#include "hip/hip_runtime.h"
#include "func.h"
/**int getindex(int x, int y)
{
	if (x > w || y > h || x < 0 || y<0)
		return 0;
	return x + y*w;
}*/

#define getindex( x,y) ( x) + ( y) * w

__global__
void Dilate_compute(uchar* dst, uchar* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int i = index % w;
    int j = index / w;

    if ( index >= h * w ) {
        return;
    }

	if ( i <= 3 || i >= w-3 || j <= 3 || j >= h-3 )
	{
        dst[index] = src[index];
        return;
    }

    int start = 0;
    for (int k = -3; k <= 3; k++)
    {
        for (int l = -3; l <= 3; l++)
        {
            if (src[getindex(i + k, j + l)] == 255)
            {
                dst[index] = 255;
                start = 1;
                break;
            }
        }
        if (start == 1)
            break;
    }
    if(start == 0)
        dst[index] = 0;
}

void Dilate(bitmap &img, uchar* &d_buffer)
{
    uchar* pixel = img.pixel;
    uchar* ptr = d_buffer;

    Dilate_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(ptr, pixel, img.h, img.w);

    img.pixel = ptr;
    d_buffer = pixel;
}


__global__
void Erode_compute(uchar* dst, uchar* src, int h, int w)
{
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    int i = index % w;
    int j = index / w;

    if ( index >= h * w ) {
        return;
    }

	if ( i <= 3 || i >= w-3 || j <= 3 || j >= h-3 )
	{
        dst[index] = src[index];
        return;
    }


    int start = 0;
    for (int k = -3; k <= 3; k++)
    {
        for (int l = -3; l <= 3; l++)
        {
            if (src[getindex(i + k, j + l)] != 255)
            {
                dst[index] = 0;
                start = 1;
                break;
            }
        }
        if (start == 1)
            break;
    }
    if (start == 0)
        dst[index] = 255;

}


void Erode(bitmap &img, uchar* &d_buffer)
{
    uchar* pixel = img.pixel;
    uchar* ptr = d_buffer;

    Erode_compute<<<(img.w * img.h + BLOCK_SIZE) / BLOCK_SIZE, BLOCK_SIZE>>>(ptr, pixel, img.h, img.w);

    img.pixel = ptr;
    d_buffer = pixel;
}
